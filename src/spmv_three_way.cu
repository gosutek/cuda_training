#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/mmio.h"
#include <algorithm>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include "../include/spmv_three_way.h"

#define BLOCK_SIZE 256
#define DEVICE_ID 0

__global__ void spmv(const CSRMatrix A, const DenseMatrix x, DenseMatrix y)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < A.rows) {

    VAL_TYPE row_result_val = 0.0;

    for (uint32_t i = A.row_ptr[row]; i < A.row_ptr[row + 1]; ++i) {
      row_result_val += A.val[i] * x.data[A.col_idx[i]];
    }
    y.data[row] = row_result_val;
  }
}

DenseMatrix spmv_l2_window(const CSRMatrix& d_A, const DenseMatrix& d_x, DenseMatrix& d_y)
{
  hipStream_t stream;
  hipStreamCreate(&stream);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, DEVICE_ID);

  size_t size = std::min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize,
                     size); // cudaLimitPersistingL2CacheSize -> Global limit for all persistent data

  size_t window_size
    = std::min(prop.accessPolicyMaxWindowSize,
               (int)d_x.data_size); // accessPolicyMaxWindowSize -> Per-stream limit for a single persistent region

  hipLaunchAttributeValue stream_attribute;
  stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void*>(d_x.data);
  stream_attribute.accessPolicyWindow.num_bytes = window_size;
  stream_attribute.accessPolicyWindow.hitRatio = 0.6;
  stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
  stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;

  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);

  dim3 dimBlock(BLOCK_SIZE);

  dim3 dimGrid((d_A.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

  spmv<<<dimGrid, dimBlock>>>(d_A, d_x, d_y);

  // Reset L2 Access to Normal
  stream_attribute.accessPolicyWindow.num_bytes = 0;
  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
  cudaCtxResetPersistingL2Cache();

  // Allocate host y
  DenseMatrix y = DenseMatrix(d_x.rows, d_x.cols);

  // Copy y to host
  hipMemcpy(y.data, d_y.data, d_y.data_size, hipMemcpyDeviceToHost);

  return y;
}

DenseMatrix spmv_global(const CSRMatrix& d_A, const DenseMatrix& d_x, DenseMatrix& d_y)
{

  dim3 dimBlock(BLOCK_SIZE);

  dim3 dimGrid((d_A.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

  spmv<<<dimGrid, dimBlock>>>(d_A, d_x, d_y);

  // Allocate host y
  DenseMatrix y = DenseMatrix(d_x.rows, d_x.cols);

  // Copy y to host
  hipMemcpy(y.data, d_y.data, d_y.data_size, hipMemcpyDeviceToHost);

  return y;
}

CSRMatrix parse_sparse_matrix(const char* filename)
{
  FILE* f;
  f = fopen(filename, "r");
  if (f == NULL) { throw std::runtime_error("Failed to open file"); }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) { throw std::runtime_error("Couldn't parse matrix"); }

  if (!mm_is_sparse(matcode)) { throw std::runtime_error("CSRMatrix is non-sparse -> should be sparse"); }

  int rows, cols, nnz;
  if (mm_read_mtx_crd_size(f, &rows, &cols, &nnz) != 0) { throw std::runtime_error("Failed to read matrix size"); }

  CSRMatrix matrix = CSRMatrix(rows, cols, nnz);

  std::vector<COO_Element> coo_elements;
  coo_elements.reserve(matrix.nnz);
  for (int i = 0; i < matrix.nnz; ++i) {
    COO_Element e;
    fscanf(f, "%u %u %lg\n", &e.row, &e.col, &e.val);
    e.row--;
    e.col--;
    coo_elements.push_back(e);
  }
  fclose(f);

  std::sort(coo_elements.begin(), coo_elements.end(),
            [](const auto& a, const auto& b) { return std::tie(a.row, a.col) < std::tie(b.row, b.col); });

  for (size_t i = 0; i < coo_elements.size(); ++i) {
    const auto& e = coo_elements[i];
    matrix.row_ptr[e.row + 1]++; // Account for element ONLY on the previous row
    matrix.col_idx[i] = e.col;
    matrix.val[i] = e.val;
  }
  std::partial_sum(matrix.row_ptr, matrix.row_ptr + (matrix.rows + 1), matrix.row_ptr);

  return matrix;
}

DenseMatrix parse_dense_matrix(const char* filename)
{
  FILE* f;
  f = fopen(filename, "r");
  if (f == NULL) { throw std::runtime_error("Failed to open file"); }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) { throw std::runtime_error("Couldn't parse matrix"); }

  if (!mm_is_dense(matcode)) { throw std::runtime_error("CSRMatrix is non-dense -> should be dense"); }

  int rows, cols;
  if (mm_read_mtx_array_size(f, &rows, &cols) != 0) { throw std::runtime_error("Failed to read matrix size"); }

  DenseMatrix matrix = DenseMatrix(rows, cols);

  for (int i = 0; i < matrix.rows; ++i) {
    VAL_TYPE e;
    fscanf(f, "%lg\n", &e);

    matrix.data[i] = e;
  }

  fclose(f);

  return matrix;
}

int main()
{

  try {

    CSRMatrix A = parse_sparse_matrix("data/scircuit.mtx");
    DenseMatrix x = parse_dense_matrix("data/scircuit_b.mtx");

    // ================================ DEVICE ALLOCATION ================================
    CSRMatrix d_A(A.rows, A.cols, A.nnz, CREATE_FOR_DEVICE);

    hipMalloc(&d_A.col_idx, d_A.col_idx_size);
    hipMalloc(&d_A.row_ptr, d_A.row_ptr_size);
    hipMalloc(&d_A.val, d_A.val_size);

    hipMemcpy(d_A.col_idx, A.col_idx, d_A.col_idx_size, hipMemcpyHostToDevice);
    hipMemcpy(d_A.row_ptr, A.row_ptr, d_A.row_ptr_size, hipMemcpyHostToDevice);
    hipMemcpy(d_A.val, A.val, d_A.val_size, hipMemcpyHostToDevice);

    DenseMatrix d_x(x.rows, x.cols, CREATE_FOR_DEVICE);

    hipMalloc(&d_x.data, d_x.data_size);

    hipMemcpy(d_x.data, x.data, d_x.data_size, hipMemcpyHostToDevice);

    DenseMatrix d_y(x.rows, x.cols, CREATE_FOR_DEVICE);

    hipMalloc(&d_y.data, d_y.data_size);

    // ================================ KERNEL EXECUTION ================================
    DenseMatrix y_global = spmv_global(d_A, d_x, d_y);
    DenseMatrix y_l2_window = spmv_l2_window(d_A, d_x, d_y);

    std::cout << y_global.data[0] << std::endl;
    std::cout << y_l2_window.data[0] << std::endl;

    // ================================ DEVICE DE-ALLOCATION ================================
    hipFree(d_A.col_idx);
    hipFree(d_A.row_ptr);
    hipFree(d_A.val);

    hipFree(d_x.data);

    hipFree(d_y.data);

  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }
}
