#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/mmio.h"
#include <algorithm>
#include <cstdint>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include <vector>

#define BLOCK_SIZE 3
#define VAL_TYPE double

struct COO_Element {
  uint32_t row, col;
  VAL_TYPE val;
};

struct CSRMatrix {
  int rows = 0, cols = 0, nnz = 0;
  std::vector<uint32_t> col_idx;
  std::vector<uint32_t> row_ptr;
  std::vector<VAL_TYPE> val;
};

/*
__global__ void spmv(CSRMatrix A, CSRMatrix x, CSRMatrix rs)
{

  float rs_value = 0;

  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < A.rows) {

    for (int e = 0; e < A.cols; ++e) rs_value += A.elements[row * A.cols + e] * x.elements[e];

    rs.elements[row] = rs_value;
  }
}
*/
void spmv_global(const CSRMatrix A, const CSRMatrix x, const CSRMatrix rs)
{

  // Load sparse

  CSRMatrix d_A;

  d_A.cols = A.cols;

  d_A.rows = A.rows;

  uint32_t* col_idx_ptr = d_A.col_idx.data();
  uint32_t* row_ptr_ptr = d_A.row_ptr.data();
  VAL_TYPE* val_ptr = d_A.val.data();

  // Allocate for the 3 arrays
  hipMalloc(&col_idx_ptr, A.col_idx.size() * sizeof(uint32_t));
  hipMalloc(&row_ptr_ptr, A.row_ptr.size() * sizeof(uint32_t));
  hipMalloc(&val_ptr, A.val.size() * sizeof(VAL_TYPE));

  // Copy 3 arrays to device
  hipMemcpy(col_idx_ptr, A.col_idx.data(), A.col_idx.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(row_ptr_ptr, A.row_ptr.data(), A.row_ptr.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(val_ptr, A.val.data(), A.val.size() * sizeof(VAL_TYPE), hipMemcpyHostToDevice);

  // Load vector
  // Allocate result
  // dim3 dimBlock(BLOCK_SIZE);

  // dim3 dimGrid((A.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

  // spmv<<<dimGrid, dimBlock>>>(d_A, d_x, d_rs);

  // Copy result to host
  // hipMemcpy(rs.elements, d_rs.elements, size, hipMemcpyDeviceToHost);

  // Deallocate A
  hipFree(col_idx_ptr);
  hipFree(row_ptr_ptr);
  hipFree(val_ptr);

  // Deallocate x

  // Deallocate result
}

void convert_coo_to_csr(std::vector<COO_Element> coo, CSRMatrix& matrix)
{
  std::sort(coo.begin(), coo.end(),
            [](const auto& a, const auto& b) { return std::tie(a.row, a.col) < std::tie(b.row, b.col); });

  matrix.col_idx.resize(matrix.nnz);
  matrix.val.resize(matrix.nnz);
  matrix.row_ptr.resize(matrix.rows + 1, 0);

  for (size_t i = 0; i < coo.size(); ++i) {
    const auto& e = coo[i];
    matrix.row_ptr[e.row + 1]++; // Account for element ONLY on the previous row
    matrix.col_idx[i] = e.col;
    matrix.val[i] = e.val;
  }
  std::partial_sum(coo.begin(), coo.end(), coo.begin());
}

CSRMatrix parse_and_convert(const char* filename)
{
  FILE* f;
  if (!(f = fopen(filename, "r"))) {
    fclose(f);
    throw std::runtime_error("Failed to open file");
  }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) {
    fclose(f);
    throw std::runtime_error("Couldn't parse matrix");
  }

  if (!mm_is_sparse(matcode)) {
    fclose(f);
    throw std::runtime_error("CSRMatrix is non-sparse");
  }

  CSRMatrix matrix;
  if (mm_read_mtx_crd_size(f, &matrix.rows, &matrix.cols, &matrix.nnz) != 0) {
    fclose(f);
    throw std::runtime_error("Failed to read matrix size");
  }

  std::vector<COO_Element> coo_elements;
  coo_elements.reserve(matrix.nnz);
  for (int i = 0; i < matrix.nnz; ++i) {
    COO_Element e;
    fscanf(f, "%u %u %lg\n", &e.row, &e.col, &e.val);
    e.row--;
    e.col--;
    coo_elements.push_back(e);
  }
  fclose(f);

  convert_coo_to_csr(std::move(coo_elements),
                     matrix); // coo_elements no longer exists
  return matrix;
}

// Driver
int main()
{
  // spmv_global(A, x, rs);

  try {
    CSRMatrix A = parse_and_convert("data/scircuit.mtx");
  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }
}
