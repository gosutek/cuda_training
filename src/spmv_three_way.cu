#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/mmio.h"
#include <algorithm>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include "../include/spmv_three_way.h"

#define BLOCK_SIZE 256

__global__ void spmv(int rows, const uint32_t* row_ptr, const uint32_t* col_idx, const VAL_TYPE* val,
                     const VAL_TYPE* x_data, VAL_TYPE* y_data)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows) {

    VAL_TYPE row_result_val = 0.0;

    for (uint32_t i = row_ptr[row]; i < row_ptr[row + 1]; ++i) { row_result_val += val[i] * x_data[col_idx[i]]; }
    y_data[row] = row_result_val;
  }
}

DenseMatrix spmv_global(const CSRMatrix& A, const DenseMatrix& x)
{

  // Load A
  CSRMatrix d_A(A.rows, A.cols, A.nnz, CREATE_FOR_DEVICE);

  // Allocate for the 3 arrays
  hipMalloc(&d_A.col_idx, A.nnz * sizeof(uint32_t));
  hipMalloc(&d_A.row_ptr, (A.rows + 1) * sizeof(uint32_t));
  hipMalloc(&d_A.val, A.nnz * sizeof(VAL_TYPE));

  // Copy 3 arrays to device
  hipMemcpy(d_A.col_idx, A.col_idx, A.nnz * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_A.row_ptr, A.row_ptr, (A.rows + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_A.val, A.val, A.nnz * sizeof(VAL_TYPE), hipMemcpyHostToDevice);
  // Load x
  DenseMatrix d_x(x.rows, x.cols, CREATE_FOR_DEVICE);

  hipMalloc(&d_x.data, (x.rows * x.cols) * sizeof(VAL_TYPE));

  hipMemcpy(d_x.data, x.data, (x.rows * x.cols) * sizeof(VAL_TYPE), hipMemcpyHostToDevice);

  // Allocate y
  DenseMatrix d_y(x.rows, x.cols, CREATE_FOR_DEVICE);

  hipMalloc(&d_y.data, (x.rows * x.cols) * sizeof(VAL_TYPE));

  dim3 dimBlock(BLOCK_SIZE);

  dim3 dimGrid((A.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

  spmv<<<dimGrid, dimBlock>>>(d_A.rows, d_A.row_ptr, d_A.col_idx, d_A.val, d_x.data, d_y.data);

  // Allocate host y
  DenseMatrix y = DenseMatrix(x.rows, x.cols);

  // Copy y to host
  hipMemcpy(y.data, d_y.data, (y.rows * y.cols) * sizeof(VAL_TYPE), hipMemcpyDeviceToHost);

  // Deallocate A
  hipFree(d_A.col_idx);
  hipFree(d_A.row_ptr);
  hipFree(d_A.val);

  // Deallocate x
  hipFree(d_x.data);

  // Deallocate y
  hipFree(d_y.data);

  return y;
}

CSRMatrix parse_sparse_matrix(const char* filename)
{
  FILE* f;
  f = fopen(filename, "r");
  if (f == NULL) { throw std::runtime_error("Failed to open file"); }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) { throw std::runtime_error("Couldn't parse matrix"); }

  if (!mm_is_sparse(matcode)) { throw std::runtime_error("CSRMatrix is non-sparse -> should be sparse"); }

  int rows, cols, nnz;
  if (mm_read_mtx_crd_size(f, &rows, &cols, &nnz) != 0) { throw std::runtime_error("Failed to read matrix size"); }

  CSRMatrix matrix = CSRMatrix(rows, cols, nnz);

  std::vector<COO_Element> coo_elements;
  coo_elements.reserve(matrix.nnz);
  for (int i = 0; i < matrix.nnz; ++i) {
    COO_Element e;
    fscanf(f, "%u %u %lg\n", &e.row, &e.col, &e.val);
    e.row--;
    e.col--;
    coo_elements.push_back(e);
  }
  fclose(f);

  std::sort(coo_elements.begin(), coo_elements.end(),
            [](const auto& a, const auto& b) { return std::tie(a.row, a.col) < std::tie(b.row, b.col); });

  for (size_t i = 0; i < coo_elements.size(); ++i) {
    const auto& e = coo_elements[i];
    matrix.row_ptr[e.row + 1]++; // Account for element ONLY on the previous row
    matrix.col_idx[i] = e.col;
    matrix.val[i] = e.val;
  }
  std::partial_sum(matrix.row_ptr, matrix.row_ptr + (matrix.rows + 1), matrix.row_ptr);

  return matrix;
}

DenseMatrix parse_dense_matrix(const char* filename)
{
  FILE* f;
  f = fopen(filename, "r");
  if (f == NULL) { throw std::runtime_error("Failed to open file"); }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) { throw std::runtime_error("Couldn't parse matrix"); }

  if (!mm_is_dense(matcode)) { throw std::runtime_error("CSRMatrix is non-dense -> should be dense"); }

  int rows, cols;
  if (mm_read_mtx_array_size(f, &rows, &cols) != 0) { throw std::runtime_error("Failed to read matrix size"); }

  DenseMatrix matrix = DenseMatrix(rows, cols);

  for (int i = 0; i < matrix.rows; ++i) {
    VAL_TYPE e;
    fscanf(f, "%lg\n", &e);

    matrix.data[i] = e;
  }

  fclose(f);

  return matrix;
}

int main()
{

  try {
    CSRMatrix A = parse_sparse_matrix("data/scircuit.mtx");
    DenseMatrix x = parse_dense_matrix("data/scircuit_b.mtx");

    DenseMatrix y_global = spmv_global(A, x);

    std::cout << y_global.data[0] << std::endl;

  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }
}
