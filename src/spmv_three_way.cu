#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/mmio.h"
#include <algorithm>
#include <cstdint>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include <vector>

#define BLOCK_SIZE 3
#define VAL_TYPE double

struct COO_Element {
  uint32_t row, col;
  VAL_TYPE val;
};

struct Matrix {
  int rows = 0, cols = 0;
};

struct CSRMatrix : public Matrix {

  int nnz = 0;

  std::vector<uint32_t> col_idx;
  std::vector<uint32_t> row_ptr;
  std::vector<VAL_TYPE> val;

  CSRMatrix(int r, int c, int z) : col_idx(z), val(z), row_ptr(r + 1)
  {
    rows = r;
    cols = c;
  }
};

struct DenseMatrix : public Matrix {
  std::vector<VAL_TYPE> data;

  DenseMatrix(uint32_t r, uint32_t c) : data(r * c, 0)
  {
    rows = r;
    cols = c;
  }
};

/*
__global__ void spmv(CSRMatrix A, CSRMatrix x, CSRMatrix rs)
{

  float rs_value = 0;

  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < A.rows) {

    for (int e = 0; e < A.cols; ++e) rs_value += A.elements[row * A.cols + e] * x.elements[e];

    rs.elements[row] = rs_value;
  }
}
*/
// void spmv_global(const CSRMatrix A, const CSRMatrix x, const CSRMatrix rs)
// {
//
//   // Load sparse
//
//   CSRMatrix d_A;
//
//   d_A.cols = A.cols;
//
//   d_A.rows = A.rows;
//
//   uint32_t* col_idx_ptr = d_A.col_idx.data();
//   uint32_t* row_ptr_ptr = d_A.row_ptr.data();
//   VAL_TYPE* val_ptr = d_A.val.data();
//
//   // Allocate for the 3 arrays
//   hipMalloc(&col_idx_ptr, A.col_idx.size() * sizeof(uint32_t));
//   hipMalloc(&row_ptr_ptr, A.row_ptr.size() * sizeof(uint32_t));
//   hipMalloc(&val_ptr, A.val.size() * sizeof(VAL_TYPE));
//
//   // Copy 3 arrays to device
//   hipMemcpy(col_idx_ptr, A.col_idx.data(), A.col_idx.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
//   hipMemcpy(row_ptr_ptr, A.row_ptr.data(), A.row_ptr.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
//   hipMemcpy(val_ptr, A.val.data(), A.val.size() * sizeof(VAL_TYPE), hipMemcpyHostToDevice);
//
//   // Load vector
//   // Allocate result
//   // dim3 dimBlock(BLOCK_SIZE);
//
//   // dim3 dimGrid((A.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
//
//   // spmv<<<dimGrid, dimBlock>>>(d_A, d_x, d_rs);
//
//   // Copy result to host
//   // hipMemcpy(rs.elements, d_rs.elements, size, hipMemcpyDeviceToHost);
//
//   // Deallocate A
//   hipFree(col_idx_ptr);
//   hipFree(row_ptr_ptr);
//   hipFree(val_ptr);
//
//   // Deallocate x
//
//   // Deallocate result
// }

std::unique_ptr<CSRMatrix> parse_sparse_matrix(const char* filename)
{
  FILE* f;
  if (!(f = fopen(filename, "r"))) {
    fclose(f);
    throw std::runtime_error("Failed to open file");
  }

  MM_typecode matcode;
  if (mm_read_banner(f, &matcode) != 0) {
    fclose(f);
    throw std::runtime_error("Couldn't parse matrix");
  }

  if (!mm_is_sparse(matcode)) {
    fclose(f);
    throw std::runtime_error("CSRMatrix is non-sparse");
  }

  CSRMatrix matrix;
  if (mm_read_mtx_crd_size(f, &matrix.rows, &matrix.cols, &matrix.nnz) != 0) {
    fclose(f);
    throw std::runtime_error("Failed to read matrix size");
  }

  std::vector<COO_Element> coo_elements;
  coo_elements.reserve(matrix->nnz);
  for (int i = 0; i < matrix->nnz; ++i) {
    COO_Element e;
    fscanf(f, "%u %u %lg\n", &e.row, &e.col, &e.val);
    e.row--;
    e.col--;
    coo_elements.push_back(e);
  }
  fclose(f);

  convert_coo_to_csr(std::move(coo_elements),
                     matrix); // coo_elements no longer exists
  return matrix;
}

// Driver
int main()
{
  // spmv_global(A, x, rs);

  try {
    CSRMatrix A = parse_and_convert("data/scircuit.mtx");
  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }
}
